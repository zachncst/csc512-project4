#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define BLOCK_SIZE 16
#define GRID_SIZE 160
#define SIZE BLOCK_SIZE*BLOCK_SIZE*GRID_SIZE*GRID_SIZE

void checkresult(float *ref, float *in, float *out, float *mul, int width){
	
	for(int i = 0 ; i < GRID_SIZE; i++){
		for(int j = 0; j < GRID_SIZE; j++){
			float sum = 0.0f;
			int start = j * BLOCK_SIZE * width + i * BLOCK_SIZE;
			for(int ii = 0; ii < BLOCK_SIZE; ii++){
				for(int jj = 0; jj < BLOCK_SIZE; jj++){
					sum += in[start + ii * width + jj] * mul[jj];
				}
			}
			for(int ii = 0; ii < BLOCK_SIZE; ii++){
				for(int jj = 0; jj < BLOCK_SIZE; jj++){
					if(jj % 2 == 0 && ii % 2 == 0)
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = 2.0 * in[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii]/sum;
					else if(jj % 2 == 1 && ii % 2 == 0)
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = in[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii]/sum;
					else if(jj % 2 == 1 && ii % 2 == 1)
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = (-1.0) * in[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii]/sum;
					else
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = 0.0f;
				}
			}
		}
	}

	for(int i = 0; i < SIZE; i++){
		if(abs(ref[i]-out[i]) > 1.e-6){
			printf("results checking failed at %d ref %f out %f\n", i, ref[i], out[i]);
			return;
		}
	}
	printf("results checking passed!\n");
}

__global__ void norm(float *in, float *out, float *mul, int width){
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

	if(tx >= width || ty >= SIZE/width) return;
	int start = blockIdx.x * blockDim.x * width + blockIdx.y * blockDim.y;
	float sum = 0.0f;

	for(int i = 0; i < BLOCK_SIZE; i++){
		for(int j = 0; j < BLOCK_SIZE; j++){
			sum += in[start + i * width + j] * mul[j];
		}
	}
	if(tx % 2 == 0 && ty % 2 == 0)
		out[tx * width + ty] = 2.0 * in[tx * width + ty]/sum;
	else if(tx % 2 == 1 && ty % 2 == 0)
		out[tx * width + ty] = in[tx * width + ty]/sum;
	else if(tx % 2 == 1 && ty % 2 == 1)
		out[tx * width + ty] = (-1.0) * in[tx * width + ty]/sum;
	else
		out[tx * width + ty] = 0.0f;

}



int main(){
	float *hA_in = (float *)malloc(SIZE * sizeof(float));
	float *hA_out = (float *)malloc(SIZE * sizeof(float));
	float *hB_in = (float *)malloc(BLOCK_SIZE * sizeof(float));
	float *ref = (float *)malloc(SIZE * sizeof(float));
	float *dA_in, *dA_out, *dB_in;

	srand(2016);

	for(int i = 0; i < SIZE; i++){
		hA_in[i] = (float)rand()/(float)RAND_MAX;
	}
	for(int i = 0; i < BLOCK_SIZE; i++){
		hB_in[i] = (float)rand()/(float)RAND_MAX;
	}

	hipMalloc((void **)&dA_in, SIZE * sizeof(float));
	hipMalloc((void **)&dA_out, SIZE * sizeof(float));
	hipMalloc((void **)&dB_in, BLOCK_SIZE * sizeof(float));

	hipMemcpy(dA_in, hA_in, SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dB_in, hB_in, BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice);
	struct timespec start, end;	
	dim3 grid(GRID_SIZE, GRID_SIZE, 1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_REALTIME, &start);

	norm<<<grid, block>>>(dA_in, dA_out, dB_in, BLOCK_SIZE * GRID_SIZE);

	hipDeviceSynchronize();
	clock_gettime(CLOCK_REALTIME, &end);

	printf("kernel time %fs\n", end.tv_sec - start.tv_sec + (end.tv_nsec - start.tv_nsec)/1.e9);
	hipMemcpy(hA_out, dA_out, SIZE * sizeof(float), hipMemcpyDeviceToHost);
	checkresult(ref, hA_in, hA_out, hB_in, BLOCK_SIZE * GRID_SIZE);

}
