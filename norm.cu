#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <cmath>
#include ""

// #include<sys/time.h>

#define BLOCK_SIZE 16
#define GRID_SIZE 128
#define SIZE BLOCK_SIZE*BLOCK_SIZE*GRID_SIZE*GRID_SIZE

void checkresult(float *ref, float *in, float *out, float *mul, int width) {
	for (int i = 0; i < GRID_SIZE; i++) {
		for (int j = 0; j < GRID_SIZE; j++) {
			float sum = 0.0f;
			int start = j * BLOCK_SIZE * width + i * BLOCK_SIZE;
			for (int ii = 0; ii < BLOCK_SIZE; ii++) {
				for (int jj = 0; jj < BLOCK_SIZE; jj++) {
					sum += in[start + ii * width + jj] * mul[jj];
				}
			}
			for (int ii = 0; ii < BLOCK_SIZE; ii++) {
				for (int jj = 0; jj < BLOCK_SIZE; jj++) {
					if (jj % 2 == 0 && ii % 2 == 0)
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = 2.0 * in[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] / sum;
					else if (jj % 2 == 1 && ii % 2 == 0)
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = in[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] / sum;
					else if (jj % 2 == 1 && ii % 2 == 1)
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = (-1.0) * in[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] / sum;
					else
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = 0.0f;
				}
			}
		}
	}

	for (int i = 0; i < SIZE; i++) {
		if (abs(ref[i] - out[i]) > 1.e-6) {
			printf("Diff %f\n", abs(ref[i] - out[i]));
			printf("results checking failed at %d ref %f out %f\n", i, ref[i], out[i]);
			return;
		}
	}
	printf("results checking passed!\n");
}

__global__ void norm(float *in, float *out, float *mul, int width) {
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

	if (tx >= width || ty >= SIZE / width) return;
	int start = blockIdx.x * blockDim.x * width + blockIdx.y * blockDim.y;
	float sum = 0.0f;

	for (int i = 0; i < BLOCK_SIZE; i++) {
		for (int j = 0; j < BLOCK_SIZE; j++) {
			sum += in[start + i * width + j] * mul[j];
		}
	}

	if (tx % 2 == 0 && ty % 2 == 0)
		out[tx * width + ty] = 2.0 * in[tx * width + ty] / sum;
	else if (tx % 2 == 1 && ty % 2 == 0)
		out[tx * width + ty] = in[tx * width + ty] / sum;
	else if (tx % 2 == 1 && ty % 2 == 1)
		out[tx * width + ty] = (-1.0) * in[tx * width + ty] / sum;
	else
		out[tx * width + ty] = 0.0f;
}

int main() {
	//float *hA_in = (float *)malloc(SIZE * sizeof(float));
	//float *hA_out = (float *)malloc(SIZE * sizeof(float));
	//float *hB_in = (float *)malloc(BLOCK_SIZE * sizeof(float));
	float *ref = (float *)malloc(SIZE * sizeof(float));
	float *hA_in, *hA_out, *hB_in;
	float *dA_in, *dA_out, *dB_in;

	hipHostMalloc((void**)&hA_in, SIZE * sizeof(float));
	hipHostMalloc((void**)&hA_out, SIZE * sizeof(float));
	hipHostMalloc((void**)&hB_in, BLOCK_SIZE * sizeof(float));

	srand(2016);

	for (int i = 0; i < SIZE; i++) {
		hA_in[i] = (float)rand() / (float)RAND_MAX;
	}
	for (int i = 0; i < BLOCK_SIZE; i++) {
		hB_in[i] = (float)rand() / (float)RAND_MAX;
	}

	hipMalloc((void **)&dA_in, SIZE * sizeof(float));
	hipMalloc((void **)&dA_out, SIZE * sizeof(float));
	hipMalloc((void **)&dB_in, BLOCK_SIZE * sizeof(float));

	hipMemcpy(dA_in, hA_in, SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dB_in, hB_in, BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 grid(GRID_SIZE, GRID_SIZE, 1);
	hipDeviceSynchronize();

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	norm << <grid, block >> > (dA_in, dA_out, dB_in, BLOCK_SIZE * GRID_SIZE);

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("kernel time %fs\n", milliseconds);
	hipMemcpy(hA_out, dA_out, SIZE * sizeof(float), hipMemcpyDeviceToHost);
	checkresult(ref, hA_in, hA_out, hB_in, BLOCK_SIZE * GRID_SIZE);
}
